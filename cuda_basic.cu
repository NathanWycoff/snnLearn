#include <iostream>
#include <R.h>
#include <Rinternals.h>

extern "C" void cool();

void cool() {
    double **a;
    hipMallocManaged(&a, 10 * sizeof(double **));

    int n_layers = 5;
    int *net_shape = (int *)malloc(5 * sizeof(int));
    net_shape[0] = 2;
    net_shape[1] = 1;
    net_shape[2] = 10;
    net_shape[3] = 2;
    net_shape[4] = 5;

    int t_steps = 35;

    // Stores electric potential for each layer in row major order.
    //double ***Vs = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***Vs;
    hipMallocManaged(&Vs, (n_layers-1) * sizeof(double **));
    printf(" %i ", n_layers-1);
    printf("|| %p  ||", Vs[0]);
    printf("a");
    for (int i = 0; i < n_layers-1; i++) {
        printf("b");
        double **Vsi;
        hipMallocManaged(&Vsi, net_shape[i+1] * sizeof(double*));
        Vs[i] = Vsi;
        //Vs[i] = (double **)calloc(net_shape[i+1], sizeof(double*));
        for (int j = 0; j < net_shape[i+1]; j++) {
            printf("c");
            double *Vsij;
            hipMallocManaged(&Vsij, (t_steps+1) * sizeof(double*));
            printf("c1");
            Vsij[0] = 0;// Initialize voltage at resting potential, assumed 0.
            printf("d");
            Vs[i][j] = Vsij;
            printf("e");
            //Vs[i][j] = (double *)calloc(t_steps + 1, sizeof(double));
        }
    }

}
