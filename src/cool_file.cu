
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <algorithm>
#include <unistd.h>


// NOTE: Need to compile in C++11 mode, add -std=c++11
// These should eventually be specifiable from R
#define TAU 1
#define V_THRESH 1.5
#define t_eps 0.0001
#define THREADS_PER_BLOCK 512


// Integrated Postsynaptic Kernel
__host__ __device__
double ipostkern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(TAU * (1 - exp(-dt / TAU)));
}

// Integrated refractory kernel.
__host__ __device__
double iprekern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(-V_THRESH);
}

// The inner product function, uses the standard R^n inner product.
__host__ __device__
double inner_prod(double *x, double *y, int n) {
    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += x[i] * y[i];
    }   
    return(sum);
}


__global__
void par_c_main_loop(double ***Vs, double ***ALPHA, double ***OMEGA, double **Fcal_l, int **f_count, double ***Ws, int* net_shape, int n_layers, 
        int t_steps, int l) {
    double t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < net_shape[l]; n += stride) {
        t = 0;
        for (int ti = 0; ti < t_steps; ti++) {
            // Calculate total postsynaptic contribution 
            int n_f = f_count[l][n];
            double psc = 0;
            for (int tfi = 0; tfi < n_f; tfi++) {
                double tf = Fcal_l[n][tfi];
                psc += ipostkern(t - tf);
            }
            ALPHA[l][ti][n] = psc;

            if (l > 0) {
                // Update refractory contribution
                n_f = f_count[l][n];
                double ref = 0;
                for (int tfi = 0; tfi < n_f; tfi++) {
                    double tf = Fcal_l[n][tfi];
                    ref += iprekern(t - tf);
                }
                OMEGA[l-1][n][ti] = ref;

                // Update potential
                Vs[l-1][n][ti+1] = inner_prod(Ws[l-1][n], ALPHA[l-1][ti], net_shape[l-1]) + OMEGA[l-1][n][ti];
                //printf("l = %d, n = %d, ti = %d", l, n, ti);
                //printf("Vsl = %d, n = %d, ti = %d", l, n, ti);

                // Check for firing neurons
                if (Vs[l-1][n][ti+1] > V_THRESH) {
                    Fcal_l[n][f_count[l][n]] = t + t_eps;
                    f_count[l][n]++;
                }
            }
            t += t_eps;
        }
    }
}

// The main simulation, using armadillo for matrix multiplication, and organized in such a way that we solve a sequence embarassingly parallelizable problems.
double **par_sim_body_c(int *net_shape, int n_layers,
        double **Fin, int *f_count_in, long long int **f_max, double ***Ws,
        int** f_count) {
    // Do simulation
    int t_steps = 35000;

    // Get the layer with the most neurons
    int max_neur = 0;
    for (int l = 0; l < n_layers; l++) {
        if (max_neur < net_shape[l]) {
            max_neur = net_shape[l];
        }
    }

    // Stores electric potential for each layer in row major order.
    //double ***Vs = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***Vs;
    hipMallocManaged(&Vs, (n_layers-1) * sizeof(double **));
    for (int i = 0; i < n_layers-1; i++) {
        double **Vsi;
        hipMallocManaged(&Vsi, net_shape[i+1] * sizeof(double*));
        Vs[i] = Vsi;
        //Vs[i] = (double **)calloc(net_shape[i+1], sizeof(double*));
        for (int j = 0; j < net_shape[i+1]; j++) {
            double *Vsij;
            hipMallocManaged(&Vsij, (t_steps+1) * sizeof(double*));
            Vsij[0] = 0;// Initialize voltage at resting potential, assumed 0.
            Vs[i][j] = Vsij;
            //Vs[i][j] = (double *)calloc(t_steps + 1, sizeof(double));
        }
    }

    printf("After Vs\n");

    // ALPHA stores integrated postsynaptic potential in column major order.
    // OMEGA stores integrated refractory contribution in row major order.
    //double ***ALPHA = (double ***)calloc(n_layers, sizeof(double**));
    //double ***OMEGA = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***ALPHA, ***OMEGA;
    hipMallocManaged(&ALPHA, n_layers * sizeof(double**));
    hipMallocManaged(&OMEGA, (n_layers-1) * sizeof(double**));
    for (int i = 0; i < n_layers; i++) {
        double **ALPHAi;
        hipMallocManaged(&ALPHAi, t_steps * sizeof(double*));
        ALPHA[i] = ALPHAi;
        //ALPHA[i] = (double **) calloc(t_steps, sizeof(double*));
        for (int j = 0; j < t_steps; j++) {
            double *ALPHAij;
            hipMallocManaged(&ALPHAij, net_shape[i] * sizeof(double));
            ALPHA[i][j] = ALPHAij;
            //ALPHA[i][j] = (double *) calloc(net_shape[i], sizeof(double));
        }
        if (i > 0) {
            double **OMEGAi;
            hipMallocManaged(&OMEGAi, net_shape[i] * sizeof(double*));
            OMEGA[i-1] = OMEGAi;
            //OMEGA[i-1] = (double **) calloc(net_shape[i], sizeof(double*));
            for (int j = 0; j < net_shape[i]; j++) {
                double *OMEGAij;
                hipMallocManaged(&OMEGAij, t_steps * sizeof(double));
                OMEGA[i-1][j] = OMEGAij;
                //OMEGA[i-1][j] = (double *) calloc(t_steps, sizeof(double));
            }
        }
    }

    printf("After ALPHA\n");

    // Storage for firing times
    //double ***u_Fcal = (double ***)calloc(n_layers, sizeof(double**));
    double ***u_Fcal;
    hipMallocManaged(&u_Fcal, n_layers * sizeof(double**));

    // Copy input spike times to unified memory.
    double **u_Fin;
    hipMallocManaged(&u_Fin, net_shape[0] * sizeof(double*));
    for (int n = 0; n < net_shape[0]; n++) {
        double *u_Finn;
        hipMallocManaged(&u_Finn, f_count_in[n] * sizeof(double));
        hipMemcpy(u_Finn, Fin[n], f_count_in[n] * sizeof(double), hipMemcpyDefault);
        u_Fin[n] = u_Finn;
    }

    printf("After inputs \n");

    //int **myarr = (int **)malloc(2*sizeof(int *));
    //myarr[0] = (int **)malloc(2*sizeof(int));
    //myarr[1] = (int **)malloc(2*sizeof(int));
    //myarr[0][0] = 0;
    //myarr[0][1] = 1;
    //myarr[1][0] = 2;
    //myarr[1][1] = 3;

    //int **d_myarr;
    //cudaMallocManaged(&d_myarr, 2*sizeof(int *));
    //cudaMemcpy(d_myarr, myarr, 2*sizeof(int *), cudaMemcpyDefault);

    int **u_f_count;
    hipMallocManaged(&u_f_count, n_layers * sizeof(int *));

    int *u_f_count_in;
    hipMallocManaged(&u_f_count_in, net_shape[0] * sizeof(int));
    hipMemcpy(u_f_count_in, f_count_in, net_shape[0] * sizeof(int), hipMemcpyDefault);

    //f_count[0] = u_f_count_in;
    hipMemcpy(&u_f_count[0], &u_f_count_in, sizeof(int *), hipMemcpyDefault);
    u_Fcal[0] = u_Fin;
    for (int l = 0; l < n_layers-1; l++) {
        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        double **Fi;
        hipMallocManaged(&Fi, net_shape[l+1] * sizeof(double *));
        u_Fcal[l+1] = Fi;

        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        int *f_countl;
        hipMallocManaged(&f_countl, net_shape[l+1] * sizeof(int));
        hipMemcpy(&u_f_count[l+1], &f_countl, sizeof(int *), hipMemcpyDefault);
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *Fln;
            hipMallocManaged(&Fln, f_max[l+1][n] * sizeof(double));
            Fi[n] = Fln;
            // Initialize storeage to -1, so any negative firing time means 
            for (int f = 0; f < f_max[l+1][n]; f++) {
                Fi[n][f] = -1;
            }
        }
    }

    printf("After Fi copy\n");

    //// Convert Connection weights to a C array
    //// Ws[i] is the ith layer, Ws[i][j] is the jth row of layer i,
    //// Ws[i][j][k] is the j,k element of layer i (row major ordering).
    //double ***Ws_c = (double***)calloc(net_size-1, sizeof(double**));
    //for (int l = 0; l < net_size-1; l++) {
    //    Ws_c[l] = (double**)calloc(net_shape[l], sizeof(double*));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l+1] * n;
    //    }
    //}



    // Copy weights to unified memory
    double ***u_Ws;
    hipMallocManaged(&u_Ws, (n_layers-1) * sizeof(double**));
    for (int l = 0; l < n_layers-1; l++) {
        double **u_Wsl;
        hipMallocManaged(&u_Wsl, (net_shape[l+1]) * sizeof(double*));
        u_Ws[l] = u_Wsl;
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *u_Wsln;
            hipMallocManaged(&u_Wsln, net_shape[l] * sizeof(double));
            hipMemcpy(u_Wsln, Ws[l][n], net_shape[l] * sizeof(double), hipMemcpyDefault);
            u_Ws[l][n] = u_Wsln;
        }
    }

    printf("After Weights copy\n");

    // Copy network shape to unified memory
    int *u_net_shape;
    hipMallocManaged(&u_net_shape, n_layers * sizeof(int));
    hipMemcpy(u_net_shape, net_shape, n_layers * sizeof(int), hipMemcpyDefault);

    // Run actual inference
    int n_blocks = max_neur / THREADS_PER_BLOCK;
    if (n_blocks == 0) {
        n_blocks = 1;
    }

    // Main Loop
    for (int l = 0; l < n_layers; l++) {
        printf(" Solving Layer %d...\n", l);
        par_c_main_loop<<<n_blocks, THREADS_PER_BLOCK>>>(Vs, ALPHA, OMEGA, u_Fcal[l], u_f_count, u_Ws, u_net_shape, n_layers, 
                t_steps, l);
        //par_c_main_loop<<<1, 1>>>(Vs, ALPHA, OMEGA, u_Fcal, u_f_count, u_Ws, u_net_shape, n_layers, 
        //        t_steps, l);
    }
    hipDeviceSynchronize();

    printf("After main loop\n");

    // Clean up
    for (int i = 0; i < n_layers-1; i++) {
        for (int j = 0; j < net_shape[i+1]; j++) {
            hipFree(Vs[i][j]); 
        }
        hipFree(Vs[i]);
    }
    hipFree(Vs);

    for (int i = 0; i < n_layers; i++) {
        for (int j = 0; j < t_steps; j++) {
            hipFree(ALPHA[i][j]);
        }
        hipFree(ALPHA[i]);
        if (i > 0) {
            for (int j = 0; j < net_shape[i]; j++) {
                hipFree(OMEGA[i-1][j]);
            }
            hipFree(OMEGA[i-1]);
        }
    }
    hipFree(ALPHA);
    hipFree(OMEGA);

    printf("After Free\n");

    // Copy Fcal to host memory
    //double ***Fcal = (double ***)malloc(n_layers * sizeof(double **));
    //for (int l = 0; l < n_layers; l++) {
    //    Fcal[l] = (double **)malloc(net_shape[l] * sizeof(double *));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Fcal[l][n] = (double *)malloc(f_max[l][n] * sizeof(double));
    //        cudaMemcpy(Fcal[l][n], u_Fcal[l][n], f_max[l][n] * sizeof(double), cudaMemcpyDefault);
    //    }
    //}
    // Copy output spikes to host memory
    double **Fout = (double **)malloc(net_shape[n_layers]*sizeof(double*));
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        Fout[n] = (double *)malloc(f_max[n_layers-1][n] * sizeof(double));
        hipMemcpy(Fout[n], u_Fcal[n_layers-1][n], f_max[n_layers-1][n] * sizeof(double), hipMemcpyDefault);
    }

    // Copy f_count to host memory
    for (int l = 0; l < n_layers; l++) {
        f_count[l] = (int *)malloc(net_shape[l] * sizeof(int));
        hipMemcpy(f_count[l], u_f_count[l], net_shape[l] * sizeof(int), hipMemcpyDefault);
    }

    printf("After ouptut spike copy\n");

    //TODO: copy f_count

    return(Fout);
}
