/** src/read_weights.c Author "Nathan Wycoff <nathanbrwycoff@gmail.com>" Date 10.26.2018  */

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <algorithm>
#include <unistd.h>


// NOTE: Need to compile in C++11 mode, add -std=c++11
// These should eventually be specifiable from R
#define TAU 1
#define V_THRESH 1.5
#define t_eps 0.001
#define THREADS_PER_BLOCK 512


// Integrated Postsynaptic Kernel
__host__ __device__
double ipostkern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(TAU * (1 - exp(-dt / TAU)));
}

// Integrated refractory kernel.
__host__ __device__
double iprekern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(-V_THRESH);
}

// The inner product function, uses the standard R^n inner product.
__host__ __device__
double inner_prod(double *x, double *y, int n) {
    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += x[i] * y[i];
    }   
    return(sum);
}


__global__
void par_c_main_loop(double ***Vs, double ***ALPHA, double ***OMEGA, double **Fcal_l, int **f_count, double ***Ws, int* net_shape, int n_layers, 
        int t_steps, int l) {
    double t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < net_shape[l]; n += stride) {
        t = 0;
        for (int ti = 0; ti < t_steps; ti++) {
            // Calculate total postsynaptic contribution 
            int n_f = f_count[l][n];
            double psc = 0;
            for (int tfi = 0; tfi < n_f; tfi++) {
                double tf = Fcal_l[n][tfi];
                psc += ipostkern(t - tf);
            }
            ALPHA[l][ti][n] = psc;

            if (l > 0) {
                // Update refractory contribution
                n_f = f_count[l][n];
                double ref = 0;
                for (int tfi = 0; tfi < n_f; tfi++) {
                    double tf = Fcal_l[n][tfi];
                    ref += iprekern(t - tf);
                }
                OMEGA[l-1][n][ti] = ref;

                // Update potential
                Vs[l-1][n][ti+1] = inner_prod(Ws[l-1][n], ALPHA[l-1][ti], net_shape[l-1]) + OMEGA[l-1][n][ti];
                //printf("l = %d, n = %d, ti = %d", l, n, ti);
                //printf("Vsl = %d, n = %d, ti = %d", l, n, ti);

                // Check for firing neurons
                if (Vs[l-1][n][ti+1] > V_THRESH) {
                    Fcal_l[n][f_count[l][n]] = t + t_eps;
                    f_count[l][n]++;
                }
            }
            t += t_eps;
        }
    }
}

// The main simulation, using armadillo for matrix multiplication, and organized in such a way that we solve a sequence embarassingly parallelizable problems.
double **par_sim_body_c(int *net_shape, int n_layers,
        double **Fin, int *f_count_in, long long int **f_max, double ***Ws,
        int** f_count) {
    // Do simulation
    int t_steps = 3500;

    // Get the layer with the most neurons
    int max_neur = 0;
    for (int l = 0; l < n_layers; l++) {
        if (max_neur < net_shape[l]) {
            max_neur = net_shape[l];
        }
    }

    // Stores electric potential for each layer in row major order.
    //double ***Vs = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***Vs;
    hipMallocManaged(&Vs, (n_layers-1) * sizeof(double **));
    for (int i = 0; i < n_layers-1; i++) {
        double **Vsi;
        hipMallocManaged(&Vsi, net_shape[i+1] * sizeof(double*));
        Vs[i] = Vsi;
        //Vs[i] = (double **)calloc(net_shape[i+1], sizeof(double*));
        for (int j = 0; j < net_shape[i+1]; j++) {
            double *Vsij;
            hipMallocManaged(&Vsij, (t_steps+1) * sizeof(double*));
            Vsij[0] = 0;// Initialize voltage at resting potential, assumed 0.
            Vs[i][j] = Vsij;
            //Vs[i][j] = (double *)calloc(t_steps + 1, sizeof(double));
        }
    }

    printf("After Vs\n");

    // ALPHA stores integrated postsynaptic potential in column major order.
    // OMEGA stores integrated refractory contribution in row major order.
    //double ***ALPHA = (double ***)calloc(n_layers, sizeof(double**));
    //double ***OMEGA = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***ALPHA, ***OMEGA;
    hipMallocManaged(&ALPHA, n_layers * sizeof(double**));
    hipMallocManaged(&OMEGA, (n_layers-1) * sizeof(double**));
    for (int i = 0; i < n_layers; i++) {
        double **ALPHAi;
        hipMallocManaged(&ALPHAi, t_steps * sizeof(double*));
        ALPHA[i] = ALPHAi;
        //ALPHA[i] = (double **) calloc(t_steps, sizeof(double*));
        for (int j = 0; j < t_steps; j++) {
            double *ALPHAij;
            hipMallocManaged(&ALPHAij, net_shape[i] * sizeof(double));
            ALPHA[i][j] = ALPHAij;
            //ALPHA[i][j] = (double *) calloc(net_shape[i], sizeof(double));
        }
        if (i > 0) {
            double **OMEGAi;
            hipMallocManaged(&OMEGAi, net_shape[i] * sizeof(double*));
            OMEGA[i-1] = OMEGAi;
            //OMEGA[i-1] = (double **) calloc(net_shape[i], sizeof(double*));
            for (int j = 0; j < net_shape[i]; j++) {
                double *OMEGAij;
                hipMallocManaged(&OMEGAij, t_steps * sizeof(double));
                OMEGA[i-1][j] = OMEGAij;
                //OMEGA[i-1][j] = (double *) calloc(t_steps, sizeof(double));
            }
        }
    }

    printf("After ALPHA\n");

    // Storage for firing times
    //double ***u_Fcal = (double ***)calloc(n_layers, sizeof(double**));
    double ***u_Fcal;
    hipMallocManaged(&u_Fcal, n_layers * sizeof(double**));

    // Copy input spike times to unified memory.
    double **u_Fin;
    hipMallocManaged(&u_Fin, net_shape[0] * sizeof(double*));
    for (int n = 0; n < net_shape[0]; n++) {
        double *u_Finn;
        hipMallocManaged(&u_Finn, f_count_in[n] * sizeof(double));
        hipMemcpy(u_Finn, Fin[n], f_count_in[n] * sizeof(double), hipMemcpyDefault);
        u_Fin[n] = u_Finn;
    }

    printf("After inputs \n");

    //int **myarr = (int **)malloc(2*sizeof(int *));
    //myarr[0] = (int **)malloc(2*sizeof(int));
    //myarr[1] = (int **)malloc(2*sizeof(int));
    //myarr[0][0] = 0;
    //myarr[0][1] = 1;
    //myarr[1][0] = 2;
    //myarr[1][1] = 3;

    //int **d_myarr;
    //cudaMallocManaged(&d_myarr, 2*sizeof(int *));
    //cudaMemcpy(d_myarr, myarr, 2*sizeof(int *), cudaMemcpyDefault);

    int **u_f_count;
    hipMallocManaged(&u_f_count, n_layers * sizeof(int *));

    int *u_f_count_in;
    hipMallocManaged(&u_f_count_in, net_shape[0] * sizeof(int));
    hipMemcpy(u_f_count_in, f_count_in, net_shape[0] * sizeof(int), hipMemcpyDefault);

    //f_count[0] = u_f_count_in;
    hipMemcpy(&u_f_count[0], &u_f_count_in, sizeof(int *), hipMemcpyDefault);
    u_Fcal[0] = u_Fin;
    for (int l = 0; l < n_layers-1; l++) {
        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        double **Fi;
        hipMallocManaged(&Fi, net_shape[l+1] * sizeof(double *));
        u_Fcal[l+1] = Fi;

        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        int *f_countl;
        hipMallocManaged(&f_countl, net_shape[l+1] * sizeof(int));
        hipMemcpy(&u_f_count[l+1], &f_countl, sizeof(int *), hipMemcpyDefault);
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *Fln;
            hipMallocManaged(&Fln, f_max[l+1][n] * sizeof(double));
            Fi[n] = Fln;
            // Initialize storeage to -1, so any negative firing time means 
            for (int f = 0; f < f_max[l+1][n]; f++) {
                Fi[n][f] = -1;
            }
        }
    }

    printf("After Fi copy\n");

    //// Convert Connection weights to a C array
    //// Ws[i] is the ith layer, Ws[i][j] is the jth row of layer i,
    //// Ws[i][j][k] is the j,k element of layer i (row major ordering).
    //double ***Ws_c = (double***)calloc(net_size-1, sizeof(double**));
    //for (int l = 0; l < net_size-1; l++) {
    //    Ws_c[l] = (double**)calloc(net_shape[l], sizeof(double*));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l+1] * n;
    //    }
    //}



    // Copy weights to unified memory
    double ***u_Ws;
    hipMallocManaged(&u_Ws, (n_layers-1) * sizeof(double**));
    for (int l = 0; l < n_layers-1; l++) {
        double **u_Wsl;
        hipMallocManaged(&u_Wsl, (net_shape[l+1]) * sizeof(double*));
        u_Ws[l] = u_Wsl;
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *u_Wsln;
            hipMallocManaged(&u_Wsln, net_shape[l] * sizeof(double));
            hipMemcpy(u_Wsln, Ws[l][n], net_shape[l] * sizeof(double), hipMemcpyDefault);
            u_Ws[l][n] = u_Wsln;
        }
    }

    printf("After Weights copy\n");

    // Copy network shape to unified memory
    int *u_net_shape;
    hipMallocManaged(&u_net_shape, n_layers * sizeof(int));
    hipMemcpy(u_net_shape, net_shape, n_layers * sizeof(int), hipMemcpyDefault);

    // Run actual inference
    int n_blocks = max_neur / THREADS_PER_BLOCK;
    if (n_blocks == 0) {
        n_blocks = 1;
    }

    for (int l = 0; l < n_layers; l++) {
        par_c_main_loop<<<n_blocks, THREADS_PER_BLOCK>>>(Vs, ALPHA, OMEGA, u_Fcal[l], u_f_count, u_Ws, u_net_shape, n_layers, 
                t_steps, l);
        //par_c_main_loop<<<1, 1>>>(Vs, ALPHA, OMEGA, u_Fcal, u_f_count, u_Ws, u_net_shape, n_layers, 
        //        t_steps, l);
    }
    hipDeviceSynchronize();

    printf("After main loop\n");

    // Clean up
    for (int i = 0; i < n_layers-1; i++) {
        for (int j = 0; j < net_shape[i+1]; j++) {
            hipFree(Vs[i][j]); 
        }
        hipFree(Vs[i]);
    }
    hipFree(Vs);

    for (int i = 0; i < n_layers; i++) {
        for (int j = 0; j < t_steps; j++) {
            hipFree(ALPHA[i][j]);
        }
        hipFree(ALPHA[i]);
        if (i > 0) {
            for (int j = 0; j < net_shape[i]; j++) {
                hipFree(OMEGA[i-1][j]);
            }
            hipFree(OMEGA[i-1]);
        }
    }
    hipFree(ALPHA);
    hipFree(OMEGA);


    // Copy Fcal to host memory
    //double ***Fcal = (double ***)malloc(n_layers * sizeof(double **));
    //for (int l = 0; l < n_layers; l++) {
    //    Fcal[l] = (double **)malloc(net_shape[l] * sizeof(double *));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Fcal[l][n] = (double *)malloc(f_max[l][n] * sizeof(double));
    //        cudaMemcpy(Fcal[l][n], u_Fcal[l][n], f_max[l][n] * sizeof(double), cudaMemcpyDefault);
    //    }
    //}
    // Copy output spikes to host memory
    double **Fout = (double **)malloc(net_shape[n_layers]*sizeof(double*));
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        Fout[n] = (double *)malloc(f_max[n_layers-1][n] * sizeof(double));
        hipMemcpy(Fout[n], u_Fcal[n_layers-1][n], f_max[n_layers-1][n] * sizeof(double), hipMemcpyDefault);
    }

    // Copy f_count to host memory
    for (int l = 0; l < n_layers; l++) {
        f_count[l] = (int *)malloc(net_shape[l] * sizeof(int));
        hipMemcpy(f_count[l], u_f_count[l], net_shape[l] * sizeof(int), hipMemcpyDefault);
    }

    //TODO: copy f_count

    return(Fout);
}

int main () {
    // Read in weight matrix and store as array.
    std::ifstream input_ws("weights_file.dat");
    
    //std::string line;
    //for (std::string line; getline(input_ws, line); ) {
    //    std::cout << line << std::endl;
    //}

    bool firstline = true;
    std::vector<int> net_shape; // Stores the size of each network layer.
    std::vector<double**> Ws;// Stores weights in col major format.
    for(std::string line; getline(input_ws, line); ) {
        // Store the network configuration
        if (firstline) {
            std::istringstream net_shape_ss(line);

            while(net_shape_ss) {
                std::string s;
                if (!getline(net_shape_ss, s, ',')) break;
                //net_shape.push_back(stoi(s));
                int x;
                sscanf(s.c_str(), "%d", &x);
                net_shape.push_back(x);
            }

            firstline = false;
        }else {
            // Get the row and column size
            int rows = (int)std::count(line.begin(), line.end(), ';') + 1;
            double **W = (double **)malloc(rows * sizeof(double *));
            std::stringstream line_stream(line);
            std::string row_string;
            int r = 0;
            while(std::getline(line_stream, row_string, ';')) {
                int rows = (int)std::count(row_string.begin(), row_string.end(), ' ') + 1;
                double *col = (double *)malloc(rows * sizeof(double));
                std::stringstream elem_stream(row_string);
                std::string elem;
                int e = 0;
                while (std::getline(elem_stream, elem, ' ')) {
                    col[e] = std::stod(elem);
                    e++;
                }
                W[r] = col;
                r++;
            }
            Ws.push_back(W);
        }
    }

    int n_layers = net_shape.size();
    //// Look through the Ws
    //printf("Ws:");
    //for (int l = 0; l < n_layers-1; l++) {
    //    for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
    //        for (int n2 = 0; n2 < net_shape[l]; n2++) {
    //            std::cout << Ws[l][n1][n2] << std::endl;
    //        }
    //    }
    //}
    
    // Read in the firing times.
    std::ifstream input_ft("input_ap.dat");

    std::vector<std::vector<double> > Fin; // Stores the size of each network layer.
    for(std::string line; std::getline(input_ft, line); ) {
        // Store the network configuration
        std::istringstream firing_times_ss(line);

        std::vector<double> Fline;
        while(firing_times_ss) {
            std::string s;
            if (!getline(firing_times_ss, s, ' ')) break;
            //Fline.push_back(stod(s));
            double x;
            std::sscanf(s.c_str(), "%lf", &x);
            Fline.push_back(x);
        }

        Fin.push_back(Fline);
    }

    //std::cout << "The Fins:" << std::endl;
    //for (int n = 0; n < Fin.size(); n++) {
    //    std::cout << "New Neuron:" << std::endl;
    //    for (int f = 0; f < Fin[n].size(); f++) {
    //        std::cout << Fin[n][f] << std::endl;
    //    }
    //}

    std::ifstream input_fmax("f_max_file.dat");
    long long int **f_max = (long long int **)malloc(n_layers * sizeof(long long int*));
    int l = 0;
    for(std::string line; std::getline(input_fmax, line, ';'); ) {
        f_max[l] = (long long int *)malloc(net_shape[l] * sizeof(long long int));
        std::stringstream line_stream(line);
        std::string neur_string;
        int n = 0;
        while(std::getline(line_stream, neur_string, ' ')) {
            f_max[l][n] = stoi(neur_string);
            n++;
        }
        l++;
    }

    // Convert input firing times to C array
    double **Fin_c = (double**)calloc(net_shape[0], sizeof(double *));
    int *f_count_in = (int *)calloc(net_shape[0], sizeof(int));
    for (int n = 0; n < net_shape[0]; n++) {
        f_count_in[n] = Fin[n].size();
        if (Fin[n].size() > 0) {
            Fin_c[n] = (double *)&Fin[n][0];
        }
    }

    // Convert Connection weights to a C array
    double ***Ws_c = (double***)calloc(net_shape.size()-1, sizeof(double**));
    for (int l = 0; l < net_shape.size()-1; l++) {
        Ws_c[l] = Ws[l];
    }

    // See if we got input firing times right
    //std::cout << "input firing times" << std::endl;
    //for (int n = 0; n < net_shape[0]; n++) {
    //    std::cout << f_count_in[n] << std::endl;
    //}

    // Do SRM0 simulation
    double **Fout;
    int **f_count = (int **)calloc(net_shape.size(), sizeof(int *));
    Fout = par_sim_body_c(&net_shape[0], net_shape.size(), Fin_c, 
            f_count_in, f_max, Ws_c, f_count);

    // Print out the results
    //for (int l = 0; l < net_shape.size(); l++) {
    //    std::cout << "Layer:" << l << std::endl;
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        for (int f = 0; f < f_count[l][n]; f++) {
    //            std::cout << "Value:" << std::endl;
    //            std::cout << Fcal[l][n][f] << std::endl;
    //        }
    //    }
    //}
    
    // Print out fire counts
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        std::cout << "Output Neuron " << n << " had " << f_count[n_layers-1][n] << " firing events" << std::endl;
    }

    //TODO: free things at some point.

    return 0;
}
