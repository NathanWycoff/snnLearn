/** src/srm0_wrapper.c Author "Nathan Wycoff <nathanbrwycoff@gmail.com>" Date 11.21.2018  */

// An R interface to srm0.cu
#include <R.h>
#include <Rinternals.h>
#include <unistd.h>
#include "srm0.h"

extern "C" void gvectorAdd(double *Ws_in, int *net_shape, int *n_layersp, double *Fin, int *f_count_in, int *f_max_R, double *Flast, int *t_steps_R, double *t_eps_R, double *gamma, double *gammad, int *debug_R, bool *copy_gamma_R);
//extern void gvectorAdd(double *Ws_in, int *net_shape, int *n_layersp, double *Fin, int *f_count_in, int *f_max_R);

void gvectorAdd(double *Ws_in, int *net_shape, int *n_layersp, double *Fin, int *f_count_in, int *f_max_R, double *Flast, int *t_steps_R, double *t_eps_R, double *gamma, double *gammad, int *debug_R, bool *copy_gamma_R) {
    // R only passes pointers.
    int n_layers = *n_layersp;
    int t_steps = *t_steps_R;
    double t_eps = *t_eps_R;
    int debug = *debug_R;
    bool copy_gamma = *copy_gamma_R;

    printf("copy_gamma: %d\n", copy_gamma);

    // Calculate the cumulative size of each layer's weight matrix
    int *wlo = (int *)calloc(n_layers-1, sizeof(int *));
    wlo[0] = 0;
    for (int l = 0; l < n_layers-2; l++) {
        wlo[l+1] = wlo[l] + net_shape[l] * net_shape[l+1];
    }

    // Convert Connection weights to a C array
    // Ws[i] is the ith layer, Ws[i][j] is the jth col of layer i,
    // Ws[i][j][k] is the k,j element of layer i (col major ordering).
    // Ws[i][j] gives all input weights to neuron j of layer i
    double ***Ws_c = (double***)calloc(n_layers-1, sizeof(double**));
    for (int l = 0; l < n_layers-1; l++) {
        Ws_c[l] = (double**)calloc(net_shape[l+1], sizeof(double*));
        for (int n = 0; n < net_shape[l+1]; n++) {
            Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l] * n;
        }
    }

    // Count number of input firings for each input neuron
    double **Fin_c = (double**)calloc(net_shape[0], sizeof(double *));
    int cum_in = 0;
    for (int n = 0; n < net_shape[0]; n++) {
        Fin_c[n] = &Fin[cum_in];
        cum_in += f_count_in[n];
    }

    int ** f_max = (int **)calloc(n_layers, sizeof(int*));
    int cum_size = 0;
    for (int l = 0; l < n_layers; l++) {
        f_max[l] = f_max_R + cum_size;
        cum_size += net_shape[l];
    }

    //// Calculate an upper bound on how many times each neuron will fire in C
    //int ** f_max = (int **)calloc(n_layers, sizeof(int*));
    //f_max[0] = f_count_in;
    //for (int l = 0; l < n_layers-1; l++) {
    //    //f_max[l+1] = conv_to<ivec>::from(TAU / V_THRESH * Ws[l].t() * f_max[l]);
    //    f_max[l+1] = (int *)calloc(net_shape[l+1], sizeof(int));
    //    for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
    //        double accum = 0;
    //        for (int n2 = 0; n2 < net_shape[l]; n2++) {
    //            accum += TAU / V_THRESH * Ws_c[l][n2][n1] * f_max[l][n2];
    //        }
    //        f_max[l+1][n1] = (int) accum;
    //    }
    //}

    // Convert to a C array for use with GPU
    long long int **f_max_c = 
        (long long int **)calloc(n_layers, sizeof(long long int*));
    for (int l = 0; l < n_layers; l++) {
        f_max_c[l] = (long long int *)calloc(net_shape[l], sizeof(long long int));
        for (int n = 0; n < net_shape[l]; n++) {
            f_max_c[l][n] = (long long int) f_max[l][n];
        }
    }

    //printf("Input validation:\n");
    //printf("n_layers: %d", n_layers);
    //printf("net_shape:\n");
    //for (int l = 0; l < n_layers; l++) {
    //    printf("l %d = %d\n", l, net_shape[l]);
    //}
    //printf("Ws: \n");
    //// Print off Ws
    //for (int l = 0; l < n_layers-1; l++) {
    //    printf("Layer %d\n", l);
    //    for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
    //        printf("Col %d\n", n1);
    //        for (int n2 = 0; n2 < net_shape[l]; n2++) {
    //            printf("Row %d\n", n2);
    //            printf("Val: %f\n", Ws_c[l][n1][n2]);
    //        }
    //    }
    //}
    //printf("Fin_c:\n");
    //for (int n = 0; n < net_shape[0]; n++) {
    //    printf("Neuron %d:\n", n);
    //    for (int f = 0; f < f_count_in[n]; f++) {
    //        printf("Firing time %d: %f\n", n, Fin_c[n][f]);
    //    }
    //}
    //printf("Scalar things:");
    //printf("n_layers: %d", n_layers);
    //printf("t_steps: %d", t_steps);
    //printf("t_eps: %f", t_eps);

    // Do SRM0 simulation
    double **Fout;
    //int **f_count = (int **)calloc(net_shape.size(), sizeof(int *));
    int **f_count;
    //double ****GAMMA, ****GAMMAd;
    double ****GAMMA = (double****)malloc((n_layers-1) * sizeof(double***));
    double ****GAMMAd = (double****)malloc((n_layers-1) * sizeof(double***));
    hipMallocManaged(&f_count, n_layers * sizeof(int *));
    Fout = par_sim_body_c(net_shape, n_layers, Fin_c, 
            f_count_in, f_max_c, Ws_c, f_count, t_steps, t_eps, GAMMA, GAMMAd, debug, copy_gamma);

    // Print out the results
    //for (int l = 0; l < n_layers; l++) {
    //    cout << "Layer:" << l << endl;
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        for (int f = 0; f < f_count[l][n]; f++) {
    //            cout << "Value:" << endl;
    //            cout << Fcal[l][n][f] << endl;
    //        }
    //    }
    //}


    //for (int l = 0; l < n_layers; l++) {
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        cout << f_max[l][n] << endl;
    //    }
    //}

    // Turn GAMMA and GAMMA_d into  flat arrays
    // d_GAMMA[on][fi][l]][[h] Gives the instantaneous postsynaptic current of neuron h of layer l to firing time fi of output neuron on.
    //for (int on = 0; on < net_shape[n_layers-1]; on++) {
    //    printf("ON: %d\n", on);
    //    for (int fi = 0; fi < f_max[n_layers-1][on]; fi++) {
    //        printf("fi: %d\n", fi);
    //        for (int l = 0; l < n_layers; l++) {
    //            printf("l: %d\n", l);
    //            for (int h = 0; h < net_shape[l]; h++) {
    //                printf("G: %f|| dG: %f\n", GAMMA[on][fi][l][h], GAMMAd[on][fi][l][h]);
    //            }
    //        }
    //    }
    //}

    printf("A");
    if (copy_gamma) {
        // Count up occurences
        int *cum_shape = (int *)calloc((n_layers+1), sizeof(int));
        for (int l = 0; l < n_layers; l++) {
            cum_shape[l+1] = net_shape[l] + cum_shape[l];
        }
        // Cumulative Firing events for the output layer
        int *fire_cum =  (int *)calloc(net_shape[n_layers-1]+1, sizeof(int));
        for (int n = 0; n < net_shape[n_layers-1]; n++) {
            fire_cum[n+1] = f_max[n_layers-1][n] + fire_cum[n];
        }

        //double *gamma = (double *)malloc(fire_cum[net_shape[n_layers-1]] * cum_shape[n_layers] * sizeof(double));
        for (int on = 0; on < net_shape[n_layers-1]; on++) {
            //printf("ON: %d\n", on);
            for (int fi = 0; fi < f_max[n_layers-1][on]; fi++) {
                //printf("fi: %d\n", fi);
                for (int l = 0; l < n_layers; l++) {
                    //printf("l: %d\n", l);
                    for (int h = 0; h < net_shape[l]; h++) {
                        //printf("G: %f|| dG: %f\n", GAMMA[on][fi][l][h], GAMMAd[on][fi][l][h]);
                        int ind = (fire_cum[on] + fi) * cum_shape[n_layers] + cum_shape[l] + h;
                        //printf("Total Capacity: %d", fire_cum[net_shape[n_layers-1]] * cum_shape[n_layers]);
                        //printf("Realized Capacity: %d", ind);
                        gamma[ind] = GAMMA[on][fi][l][h];
                        gammad[ind] = GAMMAd[on][fi][l][h];
                    }
                }
            }
        }
    }

    printf("B");

    // Print out fire counts
    if (debug > 0) {
        for (int n = 0; n < net_shape[n_layers-1]; n++) {
            printf("Output Neuron %d had %d firing events\n", n, f_count[n_layers-1][n]);
        }
    }
    printf("C");
    //Flast = (double *)calloc(cum_fires[net_shape[n_layers-1]], sizeof(double));
    int counter = 0;
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        for (int f = 0; f < f_max[n_layers-1][n]; f++) {
            Flast[counter] = Fout[n][f];
            counter++;
        }
    }

    //TODO: free things at some point.
    printf("D");
}
