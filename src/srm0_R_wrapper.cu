/** src/srm0_wrapper.c Author "Nathan Wycoff <nathanbrwycoff@gmail.com>" Date 11.21.2018  */

// An R interface to srm0.cu
#include <R.h>
#include <Rinternals.h>
#include <unistd.h>
#include "srm0.h"

extern "C" void gvectorAdd(double *Ws_in, int *net_shape, int *n_layersp, double *Fin, int *f_count_in, int *f_max_R, double *Flast, int *t_steps_R, double *t_eps_R);
//extern void gvectorAdd(double *Ws_in, int *net_shape, int *n_layersp, double *Fin, int *f_count_in, int *f_max_R);

void gvectorAdd(double *Ws_in, int *net_shape, int *n_layersp, double *Fin, int *f_count_in, int *f_max_R, double *Flast, int *t_steps_R, double *t_eps_R) {
    // R only passes pointers.
    int n_layers = *n_layersp;
    int t_steps = *t_steps_R;
    double t_eps = *t_eps_R;
    
    // Calculate the cumulative size of each layer's weight matrix
    int *wlo = (int *)calloc(n_layers-1, sizeof(int *));
    wlo[0] = 0;
    for (int l = 0; l < n_layers-2; l++) {
        wlo[l+1] = wlo[l] + net_shape[l] * net_shape[l+1];
    }

    // Convert Connection weights to a C array
    // Ws[i] is the ith layer, Ws[i][j] is the jth col of layer i,
    // Ws[i][j][k] is the k,j element of layer i (col major ordering).
    // Ws[i][j] gives all input weights to neuron j of layer i
    double ***Ws_c = (double***)calloc(n_layers-1, sizeof(double**));
    for (int l = 0; l < n_layers-1; l++) {
        Ws_c[l] = (double**)calloc(net_shape[l+1], sizeof(double*));
        for (int n = 0; n < net_shape[l+1]; n++) {
            Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l] * n;
        }
    }

    // Count number of input firings for each input neuron
    double **Fin_c = (double**)calloc(net_shape[0], sizeof(double *));
    int cum_in = 0;
    for (int n = 0; n < net_shape[0]; n++) {
        Fin_c[n] = &Fin[cum_in];
        cum_in += f_count_in[n];
    }

    int ** f_max = (int **)calloc(n_layers, sizeof(int*));
    int cum_size = 0;
    for (int l = 0; l < n_layers; l++) {
        f_max[l] = f_max_R + cum_size;
        cum_size += net_shape[l];
    }

    //// Calculate an upper bound on how many times each neuron will fire in C
    //int ** f_max = (int **)calloc(n_layers, sizeof(int*));
    //f_max[0] = f_count_in;
    //for (int l = 0; l < n_layers-1; l++) {
    //    //f_max[l+1] = conv_to<ivec>::from(TAU / V_THRESH * Ws[l].t() * f_max[l]);
    //    f_max[l+1] = (int *)calloc(net_shape[l+1], sizeof(int));
    //    for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
    //        double accum = 0;
    //        for (int n2 = 0; n2 < net_shape[l]; n2++) {
    //            accum += TAU / V_THRESH * Ws_c[l][n2][n1] * f_max[l][n2];
    //        }
    //        f_max[l+1][n1] = (int) accum;
    //    }
    //}

    // Convert to a C array for use with GPU
    long long int **f_max_c = 
        (long long int **)calloc(n_layers, sizeof(long long int*));
    for (int l = 0; l < n_layers; l++) {
        f_max_c[l] = (long long int *)calloc(net_shape[l], sizeof(long long int));
        for (int n = 0; n < net_shape[l]; n++) {
            f_max_c[l][n] = (long long int) f_max[l][n];
        }
    }

    printf("Input validation:\n");
    printf("n_layers: %d", n_layers);
    printf("net_shape:\n");
    for (int l = 0; l < n_layers; l++) {
        printf("l %d = %d\n", l, net_shape[l]);
    }
    printf("Ws: \n");
    // Print off Ws
    for (int l = 0; l < n_layers-1; l++) {
        printf("Layer %d\n", l);
        for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
            printf("Col %d\n", n1);
            for (int n2 = 0; n2 < net_shape[l]; n2++) {
                printf("Row %d\n", n2);
                printf("Val: %f\n", Ws_c[l][n1][n2]);
            }
        }
    }
    printf("Fin_c:\n");
    for (int n = 0; n < net_shape[0]; n++) {
        printf("Neuron %d:\n", n);
        for (int f = 0; f < f_count_in[n]; f++) {
            printf("Firing time %d: %f\n", n, Fin_c[n][f]);
        }
    }
    printf("Scalar things:");
    printf("n_layers: %d", n_layers);
    printf("t_steps: %d", t_steps);
    printf("t_eps: %f", t_eps);

    // Do SRM0 simulation
    double **Fout;
    //int **f_count = (int **)calloc(net_shape.size(), sizeof(int *));
    int **f_count;
    hipMallocManaged(&f_count, n_layers * sizeof(int *));
    Fout = par_sim_body_c(net_shape, n_layers, Fin_c, 
            f_count_in, f_max_c, Ws_c, f_count, t_steps, t_eps);

    // Print out the results
    //for (int l = 0; l < n_layers; l++) {
    //    cout << "Layer:" << l << endl;
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        for (int f = 0; f < f_count[l][n]; f++) {
    //            cout << "Value:" << endl;
    //            cout << Fcal[l][n][f] << endl;
    //        }
    //    }
    //}
    

    //for (int l = 0; l < n_layers; l++) {
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        cout << f_max[l][n] << endl;
    //    }
    //}

    // Print out fire counts
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        printf("Output Neuron %d had %d firing events\n", n, f_count[n_layers-1][n]);
    }

    // Pack up the last layer's firing times.
    int *cum_fires = (int *)calloc(net_shape[n_layers-1], sizeof(int));
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        if (n == 0) {
            cum_fires[n] = 0;
        } else {
            cum_fires[n] = cum_fires[n-1] + f_max[n_layers-1][n-1];
        }
    }

    //Flast = (double *)calloc(cum_fires[net_shape[n_layers-1]], sizeof(double));
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        for (int f = 0; f < f_max[n_layers-1][n]; f++) {
            printf("Assigning value %f\n", Fout[n][f]);
            Flast[f + cum_fires[n]] = Fout[n][f];
        }
    }

    //TODO: free things at some point.
}
