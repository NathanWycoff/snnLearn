/** src/srm0_wrapper.c Author "Nathan Wycoff <nathanbrwycoff@gmail.com>" Date 11.21.2018  */

// An R interface to srm0.cu
#include <R.h>
#include <Rinternals.h>
#include <unistd.h>
#include "srm0.h"

extern "C" void gvectorAdd(double *Ws_in, int *net_shape, int *net_sizep, double *Fin, int *f_count_in, int *f_max_R, double *Flast);
//extern void gvectorAdd(double *Ws_in, int *net_shape, int *net_sizep, double *Fin, int *f_count_in, int *f_max_R);

void gvectorAdd(double *Ws_in, int *net_shape, int *net_sizep, double *Fin, int *f_count_in, int *f_max_R, double *Flast) {
    // R only passes pointers.
    int net_size = *net_sizep;
    
    // Calculate the cumulative size of each layer's weight matrix
    int *wlo = (int *)calloc(net_size-1, sizeof(int *));
    wlo[0] = 0;
    for (int l = 0; l < net_size-2; l++) {
        wlo[l+1] = wlo[l] + net_shape[l] * net_shape[l+1];
    }

    // Convert Connection weights to a C array
    // Ws[i] is the ith layer, Ws[i][j] is the jth row of layer i,
    // Ws[i][j][k] is the j,k element of layer i (row major ordering).
    double ***Ws_c = (double***)calloc(net_size-1, sizeof(double**));
    for (int l = 0; l < net_size-1; l++) {
        Ws_c[l] = (double**)calloc(net_shape[l], sizeof(double*));
        for (int n = 0; n < net_shape[l]; n++) {
            Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l+1] * n;
        }
    }

    double **Fin_c = (double**)calloc(net_shape[0], sizeof(double *));
    int cum_in = 0;
    for (int n = 0; n < net_shape[0]; n++) {
        Fin_c[n] = &Fin[cum_in];
        cum_in += f_count_in[n];
    }


    int ** f_max = (int **)calloc(net_size, sizeof(int*));
    int cum_size = 0;
    for (int l = 0; l < net_size; l++) {
        f_max[l] = f_max_R + cum_size;
        cum_size += net_shape[l];
    }

    //// Calculate an upper bound on how many times each neuron will fire in C
    //int ** f_max = (int **)calloc(net_size, sizeof(int*));
    //f_max[0] = f_count_in;
    //for (int l = 0; l < net_size-1; l++) {
    //    //f_max[l+1] = conv_to<ivec>::from(TAU / V_THRESH * Ws[l].t() * f_max[l]);
    //    f_max[l+1] = (int *)calloc(net_shape[l+1], sizeof(int));
    //    for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
    //        double accum = 0;
    //        for (int n2 = 0; n2 < net_shape[l]; n2++) {
    //            accum += TAU / V_THRESH * Ws_c[l][n2][n1] * f_max[l][n2];
    //        }
    //        f_max[l+1][n1] = (int) accum;
    //    }
    //}

    // Convert to a C array for use with GPU
    long long int **f_max_c = 
        (long long int **)calloc(net_size, sizeof(long long int*));
    for (int l = 0; l < net_size; l++) {
        f_max_c[l] = (long long int *)calloc(net_shape[l], sizeof(long long int));
        for (int n = 0; n < net_shape[l]; n++) {
            f_max_c[l][n] = (long long int) f_max[l][n];
        }
    }

    // Do SRM0 simulation
    double ***Fcal;
    //int **f_count = (int **)calloc(net_shape.size(), sizeof(int *));
    int **f_count;
    hipMallocManaged(&f_count, net_size * sizeof(int *));
    Fcal = par_sim_body_c(net_shape, net_size, Fin_c, 
            f_count_in, f_max_c, Ws_c, f_count);

    // Print out the results
    //for (int l = 0; l < net_size; l++) {
    //    cout << "Layer:" << l << endl;
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        for (int f = 0; f < f_count[l][n]; f++) {
    //            cout << "Value:" << endl;
    //            cout << Fcal[l][n][f] << endl;
    //        }
    //    }
    //}
    

    //for (int l = 0; l < net_size; l++) {
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        cout << f_max[l][n] << endl;
    //    }
    //}

    // Pack up the last layer's firing times.
    int *cum_fires = (int *)calloc(net_shape[net_size-1], sizeof(int));
    for (int n = 0; n < net_shape[net_size-1]; n++) {
        if (n == 0) {
            cum_fires[n] = 0;
        } else {
            cum_fires[n] = cum_fires[n-1] + f_max[net_size-1][n-1];
        }
    }

    //Flast = (double *)calloc(cum_fires[net_shape[net_size-1]], sizeof(double));
    for (int n = 0; n < net_shape[net_size-1]; n++) {
        for (int f = 0; f < f_max[net_size-1][n]; f++) {
            Flast[f + cum_fires[n]] = Fcal[net_size-1][n][f];
        }
    }

    //TODO: free things at some point.
}
