#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <R.h>
#include <Rinternals.h>

using namespace std;

// These should eventually be specifiable from R
#define TAU 1
#define V_THRESH 1.5
#define t_eps 0.1

extern "C" void gvectorAdd(double *Ws_in, int *net_shape, int *net_sizep, double *Fin, int *f_count_in, int *f_max_R);

// Integrated Postsynaptic Kernel
__host__ __device__
double ipostkern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(TAU * (1 - exp(-dt / TAU)));
}

// Integrated refractory kernel.
__host__ __device__
double iprekern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(-V_THRESH);
}

// The inner product function, uses the standard R^n inner product.
__host__ __device__
double inner_prod(double *x, double *y, int n) {
    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += x[i] * y[i];
    }   
    return(sum);
}



__global__
void par_c_main_loop(double ***Vs, double ***ALPHA, double ***OMEGA, double ***Fcal, int **f_count, double ***Ws, int* net_shape, int n_layers, 
        int t_steps) {
    double t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int l = 0; l < n_layers; l++) {
        for (int n = index; n < net_shape[l]; n += stride) {
            t = 0;
            for (int ti = 0; ti < t_steps; ti++) {
                // Calculate total postsynaptic contribution 
                int n_f = f_count[l][n];
                double psc = 0;
                for (int tfi = 0; tfi < n_f; tfi++) {
                    double tf = Fcal[l][n][tfi];
                    psc += ipostkern(t - tf);
                }
                ALPHA[l][ti][n] = psc;

                if (l > 0) {
                    // Update refractory contribution
                    n_f = f_count[l][n];
                    double ref = 0;
                    for (int tfi = 0; tfi < n_f; tfi++) {
                        double tf = Fcal[l][n][tfi];
                        ref += iprekern(t - tf);
                    }
                    OMEGA[l-1][n][ti] = ref;

                    // Update potential
                    Vs[l-1][n][ti+1] = inner_prod(Ws[l-1][n], ALPHA[l-1][ti], net_shape[l-1]) + OMEGA[l-1][n][ti];

                    // Check for firing neurons
                    if (Vs[l-1][n][ti+1] > V_THRESH) {
                        Fcal[l][n][f_count[l][n]] = t + t_eps;
                        f_count[l][n]++;
                    }
                }
                t += t_eps;
            }
        }
    }
}

// The main simulation, using armadillo for matrix multiplication, and organized in such a way that we solve a sequence embarassingly parallelizable problems.
double ***par_sim_body_c(int *net_shape, int n_layers,
        double **Fin, int *f_count_in, long long int **f_max, double ***Ws,
        int** f_count) {
    // Do simulation
    int t_steps = 35;

    printf("333");

    // Print some info about params passed
    printf("n_layers: %d\n", n_layers);
    for (int l = 0; l < n_layers; l++) {
        printf("net_shape: %d\n", net_shape[l]);
    }
    printf("t_steps: %d\n", t_steps);

    // Stores electric potential for each layer in row major order.
    //double ***Vs = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***Vs;
    hipMallocManaged(&Vs, (n_layers-1) * sizeof(double **));
    printf(" %i ", n_layers-1);
    printf("|| %p  ||", Vs[0]);
    printf("a");
    for (int i = 0; i < n_layers-1; i++) {
        printf("b");
        double **Vsi;
        hipMallocManaged(&Vsi, net_shape[i+1] * sizeof(double*));
        Vs[i] = Vsi;
        //Vs[i] = (double **)calloc(net_shape[i+1], sizeof(double*));
        for (int j = 0; j < net_shape[i+1]; j++) {
            printf("c");
            double *Vsij;
            hipMallocManaged(&Vsij, (t_steps+1) * sizeof(double*));
            printf("c1");
            Vsij[0] = 0;// Initialize voltage at resting potential, assumed 0.
            printf("d");
            Vs[i][j] = Vsij;
            printf("e");
            //Vs[i][j] = (double *)calloc(t_steps + 1, sizeof(double));
        }
    }

    printf("AAAA");

    // ALPHA stores integrated postsynaptic potential in column major order.
    // OMEGA stores integrated refractory contribution in row major order.
    //double ***ALPHA = (double ***)calloc(n_layers, sizeof(double**));
    //double ***OMEGA = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***ALPHA, ***OMEGA;
    hipMallocManaged(&ALPHA, n_layers * sizeof(double**));
    hipMallocManaged(&OMEGA, (n_layers-1) * sizeof(double**));
    for (int i = 0; i < n_layers; i++) {
        double **ALPHAi;
        hipMallocManaged(&ALPHAi, t_steps * sizeof(double*));
        ALPHA[i] = ALPHAi;
        //ALPHA[i] = (double **) calloc(t_steps, sizeof(double*));
        for (int j = 0; j < t_steps; j++) {
            double *ALPHAij;
            hipMallocManaged(&ALPHAij, net_shape[i] * sizeof(double));
            ALPHA[i][j] = ALPHAij;
            //ALPHA[i][j] = (double *) calloc(net_shape[i], sizeof(double));
        }
        if (i > 0) {
            double **OMEGAi;
            hipMallocManaged(&OMEGAi, net_shape[i] * sizeof(double*));
            OMEGA[i-1] = OMEGAi;
            //OMEGA[i-1] = (double **) calloc(net_shape[i], sizeof(double*));
            for (int j = 0; j < net_shape[i]; j++) {
                double *OMEGAij;
                hipMallocManaged(&OMEGAij, t_steps * sizeof(double));
                OMEGA[i-1][j] = OMEGAij;
                //OMEGA[i-1][j] = (double *) calloc(t_steps, sizeof(double));
            }
        }
    }

    printf("BBBB");

    // Storage for firing times
    //double ***Fcal = (double ***)calloc(n_layers, sizeof(double**));
    double ***Fcal;
    hipMallocManaged(&Fcal, n_layers * sizeof(double**));

    // Copy input spike times to unified memory.
    double **u_Fin;
    hipMallocManaged(&u_Fin, net_shape[0] * sizeof(double*));
    for (int n = 0; n < net_shape[0]; n++) {
        double *u_Finn;
        hipMallocManaged(&u_Finn, f_count_in[n] * sizeof(double));
        hipMemcpy(u_Finn, Fin[n], net_shape[0] * sizeof(double), hipMemcpyDefault);
        u_Fin[n] = u_Finn;
    }

    printf("CCCC");

    int *u_f_count_in;
    hipMallocManaged(&u_f_count_in, net_shape[0] * sizeof(int));
    hipMemcpy(u_f_count_in, f_count_in, net_shape[0] * sizeof(int), hipMemcpyDefault);

    printf("444");

    printf("aaa");

    f_count[0] = u_f_count_in;
    Fcal[0] = u_Fin;
    for (int l = 0; l < n_layers-1; l++) {
        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        double **Fi;
        hipMallocManaged(&Fi, net_shape[l+1] * sizeof(double *));
        Fcal[l+1] = Fi;

        int *f_countl;
        hipMallocManaged(&f_countl, net_shape[l+1] * sizeof(int));
        f_count[l+1] = f_countl;
        //f_count[l+1] = (int *)calloc(net_shape[l+1], sizeof(int));
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *Fin;
            hipMallocManaged(&Fin, f_max[l+1][n] * sizeof(double));
            Fi[n] = Fin;
            // Initialize storeage to -1, so any negative firing time means 
            //Fi[n] = (double *) calloc(f_max[l+1][n], sizeof(double));
        }
    }

    printf("bbb");

    //// Convert Connection weights to a C array
    //// Ws[i] is the ith layer, Ws[i][j] is the jth row of layer i,
    //// Ws[i][j][k] is the j,k element of layer i (row major ordering).
    //double ***Ws_c = (double***)calloc(net_size-1, sizeof(double**));
    //for (int l = 0; l < net_size-1; l++) {
    //    Ws_c[l] = (double**)calloc(net_shape[l], sizeof(double*));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l+1] * n;
    //    }
    //}


    // Copy weights to unified memory
    double ***u_Ws;
    hipMallocManaged(&u_Ws, (n_layers-1) * sizeof(double**));
    for (int l = 0; l < n_layers-1; l++) {
        double **u_Wsl;
        hipMallocManaged(&u_Wsl, (net_shape[l]) * sizeof(double*));
        u_Ws[l] = u_Wsl;
        for (int n = 0; n < net_shape[l]; n++) {
            double *u_Wsln;
            hipMallocManaged(&u_Wsln, net_shape[l] * sizeof(double));
            hipMemcpy(u_Wsln, Ws[l][n], net_shape[l] * sizeof(double), hipMemcpyDefault);
            u_Ws[l][n] = u_Wsln;
        }
    }

    printf("ccc");

    // Copy network shape to unified memory
    int *u_net_shape;
    hipMallocManaged(&u_net_shape, n_layers * sizeof(int));
    hipMemcpy(u_net_shape, net_shape, n_layers * sizeof(int), hipMemcpyDefault);

    // Run actual inference
    par_c_main_loop<<<1, 1>>>(Vs, ALPHA, OMEGA, Fcal, f_count, u_Ws, u_net_shape, n_layers, 
            t_steps);

    printf("ddd");

    // Clean up
    for (int i = 0; i < n_layers-1; i++) {
        for (int j = 0; j < net_shape[i+1]; j++) {
            hipFree(Vs[i][j]); 
        }
        hipFree(Vs[i]);
    }
    hipFree(Vs);

    for (int i = 0; i < n_layers; i++) {
        for (int j = 0; j < t_steps; j++) {
            hipFree(ALPHA[i][j]);
        }
        hipFree(ALPHA[i]);
        if (i > 0) {
            for (int j = 0; j < net_shape[i]; j++) {
                hipFree(OMEGA[i-1][j]);
            }
            hipFree(OMEGA[i-1]);
        }
    }
    hipFree(ALPHA);
    hipFree(OMEGA);

    printf("555");

    return(Fcal);
}


void gvectorAdd(double *Ws_in, int *net_shape, int *net_sizep, double *Fin, int *f_count_in, int *f_max_R) {
    // R only passes pointers.
    int net_size = *net_sizep;
    
    // Calculate the cumulative size of each layer's weight matrix
    int *wlo = (int *)calloc(net_size-1, sizeof(int *));
    wlo[0] = 0;
    printf("Offsets:\n");
    printf("%i\n", wlo[0]);
    for (int l = 0; l < net_size-2; l++) {
        wlo[l+1] = wlo[l] + net_shape[l] * net_shape[l+1];
        printf("%i\n", wlo[l+1]);
    }

    // Convert Connection weights to a C array
    // Ws[i] is the ith layer, Ws[i][j] is the jth row of layer i,
    // Ws[i][j][k] is the j,k element of layer i (row major ordering).
    double ***Ws_c = (double***)calloc(net_size-1, sizeof(double**));
    for (int l = 0; l < net_size-1; l++) {
        Ws_c[l] = (double**)calloc(net_shape[l], sizeof(double*));
        for (int n = 0; n < net_shape[l]; n++) {
            Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l+1] * n;
        }
    }

    double **Fin_c = (double**)calloc(net_shape[0], sizeof(double *));
    int cum_in = 0;
    for (int n = 0; n < net_shape[0]; n++) {
        Fin_c[n] = &Fin[cum_in];
        cum_in += f_count_in[n];
    }

    printf("Firing Times:\n");
    for (int n = 0; n < net_shape[0]; n++) {
        printf("Neuron %i\n", n+1);
        for (int tf = 0; tf < f_count_in[n]; tf++) {
            printf("%f\n", Fin_c[n][tf]);
        }
    }


    int ** f_max = (int **)calloc(net_size, sizeof(int*));
    int cum_size = 0;
    for (int l = 0; l < net_size; l++) {
        f_max[l] = f_max_R + cum_size;
        cum_size += net_shape[l];
    }

    //// Calculate an upper bound on how many times each neuron will fire in C
    //int ** f_max = (int **)calloc(net_size, sizeof(int*));
    //f_max[0] = f_count_in;
    //for (int l = 0; l < net_size-1; l++) {
    //    //f_max[l+1] = conv_to<ivec>::from(TAU / V_THRESH * Ws[l].t() * f_max[l]);
    //    f_max[l+1] = (int *)calloc(net_shape[l+1], sizeof(int));
    //    for (int n1 = 0; n1 < net_shape[l+1]; n1++) {
    //        double accum = 0;
    //        for (int n2 = 0; n2 < net_shape[l]; n2++) {
    //            accum += TAU / V_THRESH * Ws_c[l][n2][n1] * f_max[l][n2];
    //        }
    //        f_max[l+1][n1] = (int) accum;
    //    }
    //}

    // Convert to a C array for use with GPU
    long long int **f_max_c = 
        (long long int **)calloc(net_size, sizeof(long long int*));
    for (int l = 0; l < net_size; l++) {
        f_max_c[l] = (long long int *)calloc(net_shape[l], sizeof(long long int));
        for (int n = 0; n < net_shape[l]; n++) {
            f_max_c[l][n] = (long long int) f_max[l][n];
        }
    }

    //printf("Max Fire Counts:\n");
    //for (int l = 0; l < net_size; l++) {
    //    printf("Layer %i\n", l);
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        printf("%i\n", f_max_c[l][n]);
    //    } 
    //}

    // Do SRM0 simulation
    double ***Fcal;
    //int **f_count = (int **)calloc(net_shape.size(), sizeof(int *));
    int **f_count;
    hipMallocManaged(&f_count, net_size * sizeof(int *));
    printf("111");
    Fcal = par_sim_body_c(net_shape, net_size, Fin_c, 
            f_count_in, f_max_c, Ws_c, f_count);
    printf("222");

    // Print out the results
    for (int l = 0; l < net_size; l++) {
        cout << "Layer:" << l << endl;
        for (int n = 0; n < net_shape[l]; n++) {
            for (int f = 0; f < f_count[l][n]; f++) {
                cout << "Value:" << endl;
                cout << Fcal[l][n][f] << endl;
            }
        }
    }
    

    for (int l = 0; l < net_size; l++) {
        for (int n = 0; n < net_shape[l]; n++) {
            cout << f_max[l][n] << endl;
        }
    }

    //TODO: free things at some point.

}
