
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <algorithm>
#include <unistd.h>


// NOTE: Need to compile in C++11 mode, add -std=c++11
// These should eventually be specifiable from R
#define TAU 1
#define V_THRESH 1.5
#define THREADS_PER_BLOCK 512


// Integrated Postsynaptic Kernel
__host__ __device__
double ipostkern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(TAU * (1 - exp(-dt / TAU)));
}

// Postsynaptic Kernel
__host__ __device__
double postkern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(exp(-dt / TAU));
}

// Postsynaptic Kernel
__host__ __device__
double dpostkern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return((-1.0) / TAU * exp(-dt / TAU));
}

// Integrated refractory kernel.
__host__ __device__
double iprekern(double dt) {
    if (dt < 0) {
        return(0);
    }
    return(-V_THRESH);
}

// The inner product function, uses the standard R^n inner product.
__host__ __device__
double inner_prod(double *x, double *y, int n) {
    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += x[i] * y[i];
    }   
    return(sum);
}

__global__
void par_c_main_loop(double ***ALPHA, double ***OMEGA, double ***Fcal, int **f_count, double ***Ws, int* net_shape, int n_layers, 
        int t_steps, double t_eps, int l, double ****GAMMA, double ****GAMMAd, const bool copy_gamma) {
    double t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < net_shape[l]; n += stride) {
        t = 0;
        for (int ti = 0; ti < t_steps; ti++) {
            // Calculate total postsynaptic contribution 
            int n_f = f_count[l][n];
            double psc = 0;
            for (int tfi = 0; tfi < n_f; tfi++) {
                double tf = Fcal[l][n][tfi];
                psc += ipostkern(t - tf);
            }
            ALPHA[l][ti][n] = psc;

            if (l > 0) {
                // Update refractory contribution
                n_f = f_count[l][n];
                double ref = 0;
                for (int tfi = 0; tfi < n_f; tfi++) {
                    double tf = Fcal[l][n][tfi];
                    ref += iprekern(t - tf);
                }
                OMEGA[l-1][n][ti] = ref;

                // Update potential
                double V_n = inner_prod(Ws[l-1][n], ALPHA[l-1][ti], net_shape[l-1]) + OMEGA[l-1][n][ti];
                //printf("l = %d, n = %d, ti = %d", l, n, ti);
                //printf("Vsl = %d, n = %d, ti = %d", l, n, ti);

                // Check for firing neurons
                if (V_n > V_THRESH) {
                    // If an output fire, record the neural state
                    if (copy_gamma && l == n_layers-1) {
                        for (int l1 = 0; l1 < n_layers; l1++) {
                            for (int h = 0; h < net_shape[l1]; h++) {
                                GAMMA[n][f_count[l][n]][l1][h] = 0;
                                GAMMAd[n][f_count[l][n]][l1][h] = 0;
                                for (int ti = 0; ti < f_count[l1][h]; ti++) {
                                    double tf = Fcal[l1][h][ti];
                                    GAMMA[n][f_count[l][n]][l1][h] += postkern(t + t_eps - tf);
                                    GAMMAd[n][f_count[l][n]][l1][h] += dpostkern(t + t_eps - tf);
                                }
                            }
                        }
                    }
                    Fcal[l][n][f_count[l][n]] = t + t_eps;
                    f_count[l][n]++;
                }
            }
            t += t_eps;
        }
    }
}


// The main simulation, using armadillo for matrix multiplication, and organized in such a way that we solve a sequence embarassingly parallelizable problems.
double **par_sim_body_c(int *net_shape, const int n_layers,
        double **Fin, int *f_count_in, long long int **f_max, double ***Ws,
        int** f_count, const int t_steps, const double t_eps, double ****GAMMA, double ****GAMMAd, const int debug, const bool copy_gamma) {

    // Get the layer with the most neurons
    int max_neur = 0;
    for (int l = 0; l < n_layers; l++) {
        if (max_neur < net_shape[l]) {
            max_neur = net_shape[l];
        }
    }

    // ALPHA stores integrated postsynaptic potential in column major order.
    // OMEGA stores integrated refractory contribution in row major order.
    //double ***ALPHA = (double ***)calloc(n_layers, sizeof(double**));
    //double ***OMEGA = (double ***)calloc(n_layers-1, sizeof(double**));
    double ***ALPHA, ***OMEGA;
    hipMallocManaged(&ALPHA, n_layers * sizeof(double**));
    hipMallocManaged(&OMEGA, (n_layers-1) * sizeof(double**));
    for (int i = 0; i < n_layers; i++) {
        double **ALPHAi;
        hipMallocManaged(&ALPHAi, t_steps * sizeof(double*));
        ALPHA[i] = ALPHAi;
        //ALPHA[i] = (double **) calloc(t_steps, sizeof(double*));
        for (int j = 0; j < t_steps; j++) {
            double *ALPHAij;
            hipMallocManaged(&ALPHAij, net_shape[i] * sizeof(double));
            ALPHA[i][j] = ALPHAij;
            //ALPHA[i][j] = (double *) calloc(net_shape[i], sizeof(double));
        }
        if (i > 0) {
            double **OMEGAi;
            hipMallocManaged(&OMEGAi, net_shape[i] * sizeof(double*));
            OMEGA[i-1] = OMEGAi;
            //OMEGA[i-1] = (double **) calloc(net_shape[i], sizeof(double*));
            for (int j = 0; j < net_shape[i]; j++) {
                double *OMEGAij;
                hipMallocManaged(&OMEGAij, t_steps * sizeof(double));
                OMEGA[i-1][j] = OMEGAij;
                //OMEGA[i-1][j] = (double *) calloc(t_steps, sizeof(double));
            }
        }
    }

    if (debug >= 1) 
        printf("After ALPHA\n");

    // Storage for firing times
    //double ***u_Fcal = (double ***)calloc(n_layers, sizeof(double**));
    double ***u_Fcal;
    hipMallocManaged(&u_Fcal, n_layers * sizeof(double**));

    // Copy input spike times to unified memory.
    double **u_Fin;
    hipMallocManaged(&u_Fin, net_shape[0] * sizeof(double*));
    for (int n = 0; n < net_shape[0]; n++) {
        double *u_Finn;
        hipMallocManaged(&u_Finn, f_count_in[n] * sizeof(double));
        hipMemcpy(u_Finn, Fin[n], f_count_in[n] * sizeof(double), hipMemcpyDefault);
        u_Fin[n] = u_Finn;
    }

    if (debug >= 1) 
        printf("After inputs \n");

    //int **myarr = (int **)malloc(2*sizeof(int *));
    //myarr[0] = (int **)malloc(2*sizeof(int));
    //myarr[1] = (int **)malloc(2*sizeof(int));
    //myarr[0][0] = 0;
    //myarr[0][1] = 1;
    //myarr[1][0] = 2;
    //myarr[1][1] = 3;

    //int **d_myarr;
    //cudaMallocManaged(&d_myarr, 2*sizeof(int *));
    //cudaMemcpy(d_myarr, myarr, 2*sizeof(int *), cudaMemcpyDefault);

    int **u_f_count;
    hipMallocManaged(&u_f_count, n_layers * sizeof(int *));

    int *u_f_count_in;
    hipMallocManaged(&u_f_count_in, net_shape[0] * sizeof(int));
    hipMemcpy(u_f_count_in, f_count_in, net_shape[0] * sizeof(int), hipMemcpyDefault);

    //f_count[0] = u_f_count_in;
    hipMemcpy(&u_f_count[0], &u_f_count_in, sizeof(int *), hipMemcpyDefault);
    u_Fcal[0] = u_Fin;
    for (int l = 0; l < n_layers-1; l++) {
        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        double **Fi;
        hipMallocManaged(&Fi, net_shape[l+1] * sizeof(double *));
        u_Fcal[l+1] = Fi;

        //double **Fi = (double **) calloc(net_shape[l+1], sizeof(double *));
        int *f_countl;
        hipMallocManaged(&f_countl, net_shape[l+1] * sizeof(int));
        hipMemcpy(&u_f_count[l+1], &f_countl, sizeof(int *), hipMemcpyDefault);
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *Fln;
            hipMallocManaged(&Fln, f_max[l+1][n] * sizeof(double));
            Fi[n] = Fln;
            // Initialize storeage to -1, so any negative firing time means 
            for (int f = 0; f < f_max[l+1][n]; f++) {
                Fi[n][f] = -1;
            }
        }
    }

    if (debug >= 1) 
        printf("After Fi copy\n");

    //// Convert Connection weights to a C array
    //// Ws[i] is the ith layer, Ws[i][j] is the jth row of layer i,
    //// Ws[i][j][k] is the j,k element of layer i (row major ordering).
    //double ***Ws_c = (double***)calloc(net_size-1, sizeof(double**));
    //for (int l = 0; l < net_size-1; l++) {
    //    Ws_c[l] = (double**)calloc(net_shape[l], sizeof(double*));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Ws_c[l][n] = Ws_in + wlo[l] + net_shape[l+1] * n;
    //    }
    //}


    // Do GAMMA(d)
    // d_GAMMA[on][fi][l]][[h] Gives the instantaneous postsynaptic current of neuron h of layer l to firing time fi of output neuron on.
    double ****d_GAMMA, ****d_GAMMAd;
    if (copy_gamma) {
        hipMallocManaged(&d_GAMMA, (n_layers-1) * sizeof(double***));
        hipMallocManaged(&d_GAMMAd, (n_layers-1) * sizeof(double***));
        for (int on = 0; on < net_shape[n_layers-1]; on++) {
            hipMallocManaged(&d_GAMMA[on], f_max[n_layers-1][on] * sizeof(double **));
            hipMallocManaged(&d_GAMMAd[on], f_max[n_layers-1][on] * sizeof(double **));
            for (int fi = 0; fi < f_max[n_layers-1][on]; fi++) {
                hipMallocManaged(&d_GAMMA[on][fi], n_layers * sizeof(double*));
                hipMallocManaged(&d_GAMMAd[on][fi], n_layers * sizeof(double*));
                for (int l = 0; l < n_layers; l++) {
                    hipMallocManaged(&d_GAMMA[on][fi][l], net_shape[l] * sizeof(double));
                    hipMallocManaged(&d_GAMMAd[on][fi][l], net_shape[l] * sizeof(double));
                    for (int h = 0; h < net_shape[l]; h++) {
                        d_GAMMA[on][fi][l][h] = -1;
                        d_GAMMAd[on][fi][l][h] = -1;
                    }
                }
            }
        }
        if (debug >= 1) 
            printf("Initted GAMMA storage \n");
    }


    // Copy weights to unified memory
    double ***u_Ws;
    hipMallocManaged(&u_Ws, (n_layers-1) * sizeof(double**));
    for (int l = 0; l < n_layers-1; l++) {
        double **u_Wsl;
        hipMallocManaged(&u_Wsl, (net_shape[l+1]) * sizeof(double*));
        u_Ws[l] = u_Wsl;
        for (int n = 0; n < net_shape[l+1]; n++) {
            double *u_Wsln;
            hipMallocManaged(&u_Wsln, net_shape[l] * sizeof(double));
            hipMemcpy(u_Wsln, Ws[l][n], net_shape[l] * sizeof(double), hipMemcpyDefault);
            u_Ws[l][n] = u_Wsln;
        }
    }

    if (debug >= 1) 
        printf("After Weights copy\n");

    // Copy network shape to unified memory
    int *u_net_shape;
    hipMallocManaged(&u_net_shape, n_layers * sizeof(int));
    hipMemcpy(u_net_shape, net_shape, n_layers * sizeof(int), hipMemcpyDefault);

    // Run actual inference
    //TODO: Should just be + 1
    int n_blocks = max_neur / THREADS_PER_BLOCK + 1;

    // Main Loop
    for (int l = 0; l < n_layers; l++) {
        if (debug >= 1) 
            printf(" Solving Layer %d...\n", l);

        par_c_main_loop<<<n_blocks, THREADS_PER_BLOCK>>>(ALPHA, OMEGA, u_Fcal, u_f_count, u_Ws, u_net_shape, n_layers, 
                t_steps, t_eps, l, d_GAMMA, d_GAMMAd, copy_gamma);

        hipDeviceSynchronize();
    }

    if (debug >= 1) 
        printf("After main loop\n");

    // Clean up
    for (int i = 0; i < n_layers; i++) {
        for (int j = 0; j < t_steps; j++) {
            hipFree(ALPHA[i][j]);
        }
        hipFree(ALPHA[i]);
        if (i > 0) {
            for (int j = 0; j < net_shape[i]; j++) {
                hipFree(OMEGA[i-1][j]);
            }
            hipFree(OMEGA[i-1]);
        }
    }
    hipFree(ALPHA);
    hipFree(OMEGA);

    if (debug >= 1) 
        printf("After Free\n");

    // Copy Fcal to host memory
    //double ***Fcal = (double ***)malloc(n_layers * sizeof(double **));
    //for (int l = 0; l < n_layers; l++) {
    //    Fcal[l] = (double **)malloc(net_shape[l] * sizeof(double *));
    //    for (int n = 0; n < net_shape[l]; n++) {
    //        Fcal[l][n] = (double *)malloc(f_max[l][n] * sizeof(double));
    //        cudaMemcpy(Fcal[l][n], u_Fcal[l][n], f_max[l][n] * sizeof(double), cudaMemcpyDefault);
    //    }
    //}
    // Copy output spikes to host memory

    double **Fout = (double **)malloc(net_shape[n_layers-1]*sizeof(double*));
    for (int n = 0; n < net_shape[n_layers-1]; n++) {
        Fout[n] = (double *)malloc(f_max[n_layers-1][n] * sizeof(double));
        hipMemcpy(Fout[n], u_Fcal[n_layers-1][n], f_max[n_layers-1][n] * sizeof(double), hipMemcpyDefault);
    }

    // Copy f_count to host memory
    for (int l = 0; l < n_layers; l++) {
        f_count[l] = (int *)malloc(net_shape[l] * sizeof(int));
        hipMemcpy(f_count[l], u_f_count[l], net_shape[l] * sizeof(int), hipMemcpyDefault);
    }

    if (debug >= 1) 
        printf("After output spike spike/f_count\n");

#if false
    // Copy to host memory
    // d_GAMMA[on][fi][l]][[h] Gives the instantaneous postsynaptic current of neuron h of layer l to firing time fi of output neuron on.
    //GAMMA = (double****)malloc((n_layers-1) * sizeof(double***));
    //GAMMAd = (double****)malloc((n_layers-1) * sizeof(double***));
    for (int on = 0; on < net_shape[n_layers-1]; on++) {
        GAMMA[on] = (double***)malloc(f_max[n_layers-1][on] * sizeof(double**));
        GAMMAd[on] = (double***)malloc(f_max[n_layers-1][on] * sizeof(double**));
        for (int fi = 0; fi < f_max[n_layers-1][on]; fi++) {
            GAMMA[on][fi] = (double**)malloc(n_layers * sizeof(double*));
            GAMMAd[on][fi] = (double**)malloc(n_layers * sizeof(double*));
            for (int l = 0; l < n_layers; l++) {
                GAMMA[on][fi][l] = (double*)malloc(net_shape[l] * sizeof(double));
                GAMMAd[on][fi][l] = (double*)malloc(net_shape[l] * sizeof(double));
                cudaMemcpy(GAMMA[on][fi][l], d_GAMMA[on][fi][l], net_shape[l] * sizeof(double), cudaMemcpyDefault);
                cudaMemcpy(GAMMAd[on][fi][l], d_GAMMAd[on][fi][l], net_shape[l] * sizeof(double), cudaMemcpyDefault);
            }
        }
    }

#endif

    if (debug >= 1) 
        printf("After GAMMA copy\n");

    //TODO: copy f_count

    return(Fout);
}
